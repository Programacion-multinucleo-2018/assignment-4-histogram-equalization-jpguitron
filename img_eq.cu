#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "common.h"
#include <hip/hip_runtime.h>

#include <chrono>
//nvcc -o exe img_eq.cu -lopencv_core -lopencv_highgui -lopencv_imgproc -std=c++11
//se requieren minimo 256 threadspor bloque
//falta hacer por si se acaban los bloques
#define N 5//Change equalizationing window size

using namespace std;



__global__ void equalization_kernel(unsigned char* input, unsigned char* output, int width, int height, int step, int *temp)
{

	//float imgSize= width * height;
	float imgSize= blockDim.x * blockDim.y;

	int x = threadIdx.x;
	int y = threadIdx.y;

	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	const int tid = yIndex*step+xIndex;

	__shared__ int h[256];
	__shared__ int h_s[256];

	int yxn = x+y*blockDim.x;

	if(yxn < 256)
	{
			h[yxn] = 0;
			h_s[yxn] = 0;
	}
	__syncthreads();
	
	if((xIndex < width) && (yIndex < height))
	{
		atomicAdd(&h[input[tid]], 1);

	}

	__syncthreads();


	if(yxn < 256)
	{
		int a = 0;
		for(int x = 0; x <= yxn; x++)
		{
			atomicAdd(&h_s[yxn], h[x]);
			a += h[x];
		}
	}
	__syncthreads();

	
	/*if(yxn < 256)
	{
		h_s[yxn] = h_s[yxn]*(255/imgSize); 	
	}

	__syncthreads();

	if((xIndex < width) && (yIndex < height))
	{
		int actual = input[tid];
		output[tid] = h_s[actual];
		
	}*/

	if(yxn < 256)
	{
		if(h_s[yxn] != 0)
			atomicAdd(&temp[yxn], h_s[yxn]);//¿condición de carrera?

		
		if(blockIdx.x ==0 && blockIdx.y==0 &&yxn < 256 &&h_s[yxn]!=0 )
		{
			printf("%d %d\n",h_s[yxn], yxn);
		}
	}
	
	__syncthreads();
	if(yxn < 256 && blockIdx.x == 0 && blockIdx.y==0)
	{

		temp[yxn] = temp[yxn]*(255/imgSize); 	

	}

	__syncthreads();

	if((xIndex < width) && (yIndex < height))
	{
		int actual = input[tid];
		output[tid] = temp[actual];
	}

}

void equalization(const cv::Mat& input, cv::Mat& output)
{

	cout << "Input image step: " << input.step << " rows: " << input.rows << " cols: " << input.cols << endl;

	size_t colorBytes = input.step * input.rows;
	size_t grayBytes = output.step * output.rows;
	size_t tempSize = 256 *sizeof(int);

	int *temp;
	unsigned char *d_input, *d_output;

	// Allocate device memory
	SAFE_CALL(hipMalloc(&d_input, colorBytes), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc(&d_output, grayBytes), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc(&temp, tempSize), "CUDA Malloc Failed");

	// Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input, input.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
	SAFE_CALL(hipMemcpy(d_output, output.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

	// Specify a reasonable block size
	const dim3 block(16, 16);

	// Calculate grid size to cover the whole image
	const dim3 grid((int)ceil((float)input.cols / block.x), (int)ceil((float)input.rows/ block.y));
	printf("equalization_kernel<<<(%d, %d) , (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);

	// Launch the color conversion kernel
	equalization_kernel <<<grid, block >>>(d_input, d_output, input.cols, input.rows, static_cast<int>(input.step),temp);

	// Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

	// Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	// Free the device memory
	SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
}

int main(int argc, char *argv[])
{
	string imagePath;

	if(argc < 2)
		imagePath = "Images/dog1.jpeg";
  	else
  		imagePath = argv[1];

	// Read input image from the disk
	cv::Mat input = cv::imread(imagePath, CV_LOAD_IMAGE_COLOR);

	if (input.empty())
	{
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	cv::Mat input_bw(input.rows, input.cols, CV_8UC1);
	cv::Mat output(input.rows, input.cols, CV_8UC1);

	cv::cvtColor(input, input_bw, cv::COLOR_BGR2GRAY);

	auto start_cpu =  chrono::high_resolution_clock::now();
	equalization(input_bw, output);
	auto end_cpu =  chrono::high_resolution_clock::now();
	chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;
	printf("elapsed %f ms\n", duration_ms.count());


	//Allow the windows to resize
	namedWindow("Input", cv::WINDOW_NORMAL);
	namedWindow("Output", cv::WINDOW_NORMAL);

	//Show the input and output
	imshow("Input", input_bw);
	imshow("Output", output);

	//Wait for key press
	cv::waitKey();

	return 0;
}
